#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wignored-attributes"
#include <iostream>
#include <chrono>
#include <time.h>
#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_int_distribution.hpp>

#define CHUNK_SIZE (1024*1024*256)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ unsigned int parity(unsigned int x) {
    unsigned int y;
    y = x ^ (x >> 1);
    y = y ^ (y >> 2);
    y = y ^ (y >> 4);
    y = y ^ (y >> 8);
    y = y ^ (y >>16);
    return y & 1;
 }
 
 __device__ unsigned int nonlinear_h(unsigned int x) {
     return parity((x >> 0) | (x >> 1));
 }
 
 __device__ unsigned int is_set(unsigned int x, unsigned int pos) {
     return (x >> pos) & 1;
 }
 

 __device__ unsigned int h3(unsigned int x, unsigned int nsketches, unsigned int* seed) {
    unsigned int hash = 0;
    for(int i = 0; i < 32; i++){
        hash ^=  seed[i*nsketches]*is_set(x,i); 
    }
    return hash;
}


 __device__ int ech3(unsigned int v, unsigned int seed, unsigned int sbit){
     //First we compute the bitwise AND between the seed and the value
     int res = parity(v & seed) ^ nonlinear_h(v) ^ sbit ;
     //Aaaand here comes the parity
     return 2*res-1;
 }


//Full version, operates row by row
 __global__ void construct_sketch_full_r(
    unsigned int r,
    unsigned int skn_cols,
    unsigned long n_values,
    unsigned int* __restrict__ key,
    unsigned int* __restrict__ value,
    unsigned int* __restrict__ select_seed,
    unsigned long long int* __restrict__ sketches_count,
    unsigned long long int* __restrict__ sketches_sum,
    unsigned int* __restrict__ sketches_min,
    unsigned int* __restrict__ sketches_max
) 
{
    unsigned int global_size = gridDim.x * blockDim.x;
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;


    for(unsigned long i = global_id; i < n_values; i += global_size){
            unsigned int select = 0;
            for(int k = 0; k < 32; k++)  if(is_set(key[i],k)) select ^= select_seed[r*32+k];
            select = select % skn_cols;

            atomicAdd(&sketches_count[r*skn_cols+select], 1UL);
            atomicAdd(&sketches_sum[r*skn_cols+select], (unsigned long) value[i]);
            atomicMin(&sketches_min[r*skn_cols+select], value[i]);
            atomicMax(&sketches_max[r*skn_cols+select], value[i]);
    }
}

//Full version
 __global__ void construct_sketch_cnt(
    unsigned int r,
    unsigned int skn_rows,
    unsigned int skn_cols,
    unsigned long n_values,
    unsigned int* __restrict__ key,
    unsigned int* __restrict__ select_seed,
    unsigned long long int* __restrict__ sketches_count,
    unsigned int* __restrict__ hashes
) 
{
    unsigned int global_size = gridDim.x * blockDim.x;
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;


    for(unsigned long i = global_id; i < n_values; i += global_size){
        unsigned int select = 0;
        for(int k = 0; k < 32; k++)  if(is_set(key[i],k)) select ^= select_seed[r*32+k];
        select = select % skn_cols;
        hashes[i] = select;

        atomicAdd(&sketches_count[r*skn_cols+select], 1UL);
    }
}

 __global__ void construct_sketch_sum(
    unsigned int r,
    unsigned int skn_rows,
    unsigned int skn_cols,
    unsigned long n_values,
    unsigned int* __restrict__ hashes,
    unsigned int* __restrict__ value,
    unsigned long long int* __restrict__ sketches_sum
) 
{
    unsigned int global_size = gridDim.x * blockDim.x;
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;


    for(unsigned long i = global_id; i < n_values; i += global_size){
            unsigned int select = hashes[i];
            atomicAdd(&sketches_sum[r*skn_cols+select], (unsigned long) value[i]);
    }
}

 __global__ void construct_sketch_max(
    unsigned int r,
    unsigned int skn_rows,
    unsigned int skn_cols,
    unsigned long n_values,
    unsigned int* __restrict__ hashes,
    unsigned int* __restrict__ value,
    unsigned long long int* __restrict__ sketches_max
) 
{
    unsigned int global_size = gridDim.x * blockDim.x;
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;


    for(unsigned long i = global_id; i < n_values; i += global_size){
            unsigned int select = hashes[i];
            atomicMax(&sketches_max[r*skn_cols+select], value[i]);
    }
}

 __global__ void construct_sketch_min(
    unsigned int r,
    unsigned int skn_rows,
    unsigned int skn_cols,
    unsigned long n_values,
    unsigned int* __restrict__ hashes,
    unsigned int* __restrict__ value,
    unsigned long long int* __restrict__ sketches_min
) 
{
    unsigned int global_size = gridDim.x * blockDim.x;
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;


    for(unsigned long i = global_id; i < n_values; i += global_size){
            unsigned int select = hashes[i];
            atomicMin(&sketches_min[r*skn_cols+select], value[i]);
    }
}

typedef struct{

    size_t skn_rows;
    size_t skn_cols;

    unsigned long long int* sk_cnt;
    unsigned long long int* sk_sum;
    unsigned int* sk_min;
    unsigned int* sk_max;

    unsigned long long int* gt_cnt;
    unsigned long long int* gt_sum;
    unsigned int* gt_min;
    unsigned int* gt_max;
    unsigned int* gt_keys;

    unsigned long ts;
    unsigned long nkeys;

    unsigned int* keys;
    unsigned int* values;
    unsigned int* select_seed;
    unsigned long long int* errors;

} parameters;

void* cudaAllocAndCopy(void* hst_ptr, size_t size){
    void* d_ptr;
    hipMalloc((void **) &d_ptr, size);
    hipMemcpy(d_ptr, hst_ptr, size, hipMemcpyHostToDevice);
    return d_ptr;
}

void writeSArrayToFile(const char* filename, int* elements, size_t size){
    FILE *f1 = fopen(filename, "w");
    assert(f1 != NULL);
    
    fwrite(elements, sizeof(int), size, f1);
    fclose(f1);
}

void* readMappedArrayFromFile(const char* filename, size_t * filesize = NULL){
    FILE *f1 = fopen(filename, "rb");
    assert(f1 != NULL);
    fseek(f1, 0, SEEK_END);
    size_t fsize1 = ftell(f1);
    if(filesize) *filesize=fsize1;
    fseek(f1, 0, SEEK_SET);
    void* tab1;
    hipHostAlloc(&tab1,fsize1, hipHostMallocMapped);
    size_t x = fread(tab1, fsize1, 1, f1);
    fclose(f1);

    return tab1;
}

void* readArrayFromFile(const char* filename, size_t * filesize = NULL){
    FILE *f1 = fopen(filename, "rb");
    assert(f1 != NULL);
    fseek(f1, 0, SEEK_END);
    size_t fsize1 = ftell(f1);
    if(filesize) *filesize=fsize1;
    fseek(f1, 0, SEEK_SET);
    void* tab1;
    void* gtab1;
    hipMalloc(&gtab1, fsize1);
    tab1 = malloc(fsize1);
    size_t x = fread(tab1, fsize1, 1, f1);
    fclose(f1);

    hipMemcpy(gtab1, tab1, fsize1, hipMemcpyHostToDevice);
    return gtab1;
}

double sketch_contruction(parameters* p){
    size_t local = 64;
    int tot_SM = 0;
    hipDeviceGetAttribute(&tot_SM, hipDeviceAttributeMultiprocessorCount, 0);

    int occupancy = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&occupancy, construct_sketch_full_r, local, 0);

    size_t global = occupancy*tot_SM*local;

    auto begin = std::chrono::high_resolution_clock::now();
    for(unsigned int r = 0; r < p->skn_rows; r++){
        construct_sketch_full_r<<<global/local, local>>>(r, (unsigned int) p->skn_cols, p->ts, p->keys, p->values, p->select_seed, p->sk_cnt, p->sk_sum, p->sk_min, p->sk_max);
        gpuErrchk(hipPeekAtLastError());
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration_cast<std::chrono::nanoseconds>(end-begin).count();
}


int main( int argc, const char* argv[] )
{
    parameters p;
    hipSetDevice(1);
    hipSetDeviceFlags(hipDeviceMapHost);

    p.skn_rows = (unsigned int) atoll(argv[1]);
    p.skn_cols = (unsigned int) atoll(argv[2]);

    
    //Initialize sketches
    hipMalloc((void **) &p.sk_cnt, p.skn_rows*p.skn_cols*sizeof(unsigned long long int));
    hipMemset(p.sk_cnt, 0, p.skn_rows*p.skn_cols*sizeof(unsigned long));

    hipMalloc((void **) &p.sk_sum, p.skn_rows*p.skn_cols*sizeof(unsigned long long int));
    hipMemset(p.sk_sum, 0, p.skn_rows*p.skn_cols*sizeof(unsigned long));

    hipMalloc((void **) &p.sk_min, p.skn_rows*p.skn_cols*sizeof(unsigned int));
    hipMemset(p.sk_min, 255, p.skn_rows*p.skn_cols*sizeof(unsigned int));

    hipMalloc((void **) &p.sk_max, p.skn_rows*p.skn_cols*sizeof(unsigned int));
    hipMemset(p.sk_max, 0, p.skn_rows*p.skn_cols*sizeof(unsigned int));

    hipMalloc((void **) &p.errors, 6*sizeof(unsigned long));
    hipMemset(p.errors, 0, 6*sizeof(unsigned long));

    size_t size = 0;
    p.keys = (unsigned int *) readArrayFromFile(argv[3], &size);
    p.values = (unsigned int *)readArrayFromFile(argv[4],&size);
    p.ts = size/sizeof(unsigned int);

    unsigned int* select_seed =  (unsigned int*) malloc(sizeof(unsigned int)*32*32*p.skn_rows);

    boost::random::mt19937 gen(time(0));
    for(unsigned int i = 0; i < p.skn_rows*32*32; i++){
       select_seed[i] = gen();
    }
    p.select_seed = (unsigned int*) cudaAllocAndCopy(select_seed, p.skn_rows*32*32*sizeof(unsigned int));

    double time = sketch_contruction(&p);
    std::cout << p.skn_rows << ";" << p.skn_cols << ";" << 0 << ";" << p.ts << ";" << p.ts*sizeof(unsigned int)*2*8 / (1000.0*1000.0*1000.0*time / 1e9) << std::endl;

    return 0;
}
