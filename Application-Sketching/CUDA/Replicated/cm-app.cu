#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wignored-attributes"
#include <iostream>
#include <chrono>
#include <time.h>
#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_int_distribution.hpp>

#define CHUNK_SIZE (1024*1024*256)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ unsigned int parity(unsigned int x) {
    unsigned int y;
    y = x ^ (x >> 1);
    y = y ^ (y >> 2);
    y = y ^ (y >> 4);
    y = y ^ (y >> 8);
    y = y ^ (y >>16);
    return y & 1;
 }
 
 __device__ unsigned int nonlinear_h(unsigned int x) {
     return parity((x >> 0) | (x >> 1));
 }
 
 __device__ unsigned int is_set(unsigned int x, unsigned int pos) {
     return (x >> pos) & 1;
 }
 

 __device__ unsigned int h3(unsigned int x, unsigned int nsketches, unsigned int* seed) {
    unsigned int hash = 0;
    for(int i = 0; i < 32; i++){
        hash ^=  seed[i*nsketches]*is_set(x,i); 
    }
    return hash;
}


 __device__ int ech3(unsigned int v, unsigned int seed, unsigned int sbit){
     //First we compute the bitwise AND between the seed and the value
     int res = parity(v & seed) ^ nonlinear_h(v) ^ sbit ;
     //Aaaand here comes the parity
     return 2*res-1;
 }


//Full version, operates row by row
 __global__ void construct_sketch_full_r(
    unsigned int skn_rows,
    unsigned int skn_cols,
    unsigned int n_replicas,
    unsigned long n_values,
    unsigned int* __restrict__ key,
    unsigned int* __restrict__ value,
    unsigned int* __restrict__ select_seed,
    unsigned long long int* __restrict__ sketches_count,
    unsigned long long int* __restrict__ sketches_sum,
    unsigned int* __restrict__ sketches_min,
    unsigned int* __restrict__ sketches_max
) 
{
    unsigned int global_size = gridDim.x * blockDim.x;
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long n_partitions = global_size / skn_rows;
    //unsigned long partition_size = (n_values-1)/n_replicas+1;
    unsigned int partition = global_id % n_partitions;
    unsigned int r = global_id / n_partitions;
    unsigned int replica = r / skn_rows;
    unsigned int row = r % skn_rows;


    for(unsigned long i = partition + (replica*n_partitions); i < n_values; i += (n_partitions*n_replicas)){
            unsigned int select = 0;
            for(int k = 0; k < 32; k++)  if(is_set(key[i],k)) select ^= select_seed[row*32+k];
            select = select % skn_cols;

            atomicAdd(&sketches_count[r*skn_cols+select], 1UL);
            atomicAdd(&sketches_sum[r*skn_cols+select], (unsigned long) value[i]);
            atomicMin(&sketches_min[r*skn_cols+select], value[i]);
            atomicMax(&sketches_max[r*skn_cols+select], value[i]);
    }
}


typedef struct{

    size_t skn_rows;
    size_t skn_cols;
    size_t replicas;

    unsigned long long int* sk_cnt;
    unsigned long long int* sk_sum;
    unsigned int* sk_min;
    unsigned int* sk_max;

    unsigned long ts;
    unsigned long nkeys;

    unsigned int* keys;
    unsigned int* values;
    unsigned int* select_seed;

} parameters;

void* cudaAllocAndCopy(void* hst_ptr, size_t size){
    void* d_ptr;
    hipMalloc((void **) &d_ptr, size);
    hipMemcpy(d_ptr, hst_ptr, size, hipMemcpyHostToDevice);
    return d_ptr;
}

void writeSArrayToFile(const char* filename, int* elements, size_t size){
    FILE *f1 = fopen(filename, "w");
    assert(f1 != NULL);
    
    fwrite(elements, sizeof(int), size, f1);
    fclose(f1);
}

void* readMappedArrayFromFile(const char* filename, size_t * filesize = NULL){
    FILE *f1 = fopen(filename, "rb");
    assert(f1 != NULL);
    fseek(f1, 0, SEEK_END);
    size_t fsize1 = ftell(f1);
    if(filesize) *filesize=fsize1;
    fseek(f1, 0, SEEK_SET);
    void* tab1;
    hipHostAlloc(&tab1,fsize1, hipHostMallocMapped);
    size_t x = fread(tab1, fsize1, 1, f1);
    fclose(f1);

    return tab1;
}

void* readArrayFromFile(const char* filename, size_t * filesize = NULL){
    FILE *f1 = fopen(filename, "rb");
    assert(f1 != NULL);
    fseek(f1, 0, SEEK_END);
    size_t fsize1 = ftell(f1);
    if(filesize) *filesize=fsize1;
    fseek(f1, 0, SEEK_SET);
    void* tab1;
    void* gtab1;
    hipMalloc(&gtab1, fsize1);
    gpuErrchk(hipPeekAtLastError());
    tab1 = malloc(fsize1);
    size_t x = fread(tab1, fsize1, 1, f1);
    fclose(f1);

    hipMemcpy(gtab1, tab1, fsize1, hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
    return gtab1;
}

double sketch_contruction(parameters* p){
    size_t local = 64;
    int tot_SM = 0;
    hipDeviceGetAttribute(&tot_SM, hipDeviceAttributeMultiprocessorCount, 0);

    int occupancy = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&occupancy, construct_sketch_full_r, local, 0);

    size_t global = occupancy*tot_SM*local;
    auto begin = std::chrono::high_resolution_clock::now();
    
    construct_sketch_full_r<<<global/local, local>>>((unsigned int) (p->skn_rows*p->replicas), (unsigned int) p->skn_cols, (unsigned int) p->replicas, p->ts, p->keys, p->values, p->select_seed, p->sk_cnt, p->sk_sum, p->sk_min, p->sk_max);
    gpuErrchk(hipPeekAtLastError());
    
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration_cast<std::chrono::nanoseconds>(end-begin).count();
}


int main( int argc, const char* argv[] )
{
    parameters p;
    hipSetDevice(1);
    hipSetDeviceFlags(hipDeviceMapHost);

    p.skn_rows = (unsigned int) atoll(argv[1]);
    p.skn_cols = (unsigned int) atoll(argv[2]);
    p.replicas = (unsigned int) atoll(argv[3]);

    
    //Initialize sketches
    hipMalloc((void **) &p.sk_cnt, p.replicas*p.skn_rows*p.skn_cols*sizeof(unsigned long long int));
    gpuErrchk(hipPeekAtLastError());
    hipMemset(p.sk_cnt, 0, p.replicas*p.skn_rows*p.skn_cols*sizeof(unsigned long));
    gpuErrchk(hipPeekAtLastError());

    hipMalloc((void **) &p.sk_sum, p.replicas*p.skn_rows*p.skn_cols*sizeof(unsigned long long int));
    gpuErrchk(hipPeekAtLastError());
    hipMemset(p.sk_sum, 0, p.replicas*p.skn_rows*p.skn_cols*sizeof(unsigned long));
    gpuErrchk(hipPeekAtLastError());

    hipMalloc((void **) &p.sk_min, p.replicas*p.skn_rows*p.skn_cols*sizeof(unsigned int));
    gpuErrchk(hipPeekAtLastError());
    hipMemset(p.sk_min, 255, p.replicas*p.skn_rows*p.skn_cols*sizeof(unsigned int));
    gpuErrchk(hipPeekAtLastError());

    hipMalloc((void **) &p.sk_max, p.replicas*p.skn_rows*p.skn_cols*sizeof(unsigned int));
    gpuErrchk(hipPeekAtLastError());
    hipMemset(p.sk_max, 0, p.replicas*p.skn_rows*p.skn_cols*sizeof(unsigned int));
    gpuErrchk(hipPeekAtLastError());

    size_t size = 0;
    p.keys = (unsigned int *) readArrayFromFile(argv[4], &size);
    p.values = (unsigned int *)readArrayFromFile(argv[5],&size);
    p.ts = size/sizeof(unsigned int);

    unsigned int* select_seed =  (unsigned int*) malloc(sizeof(unsigned int)*32*32*p.skn_rows);

    boost::random::mt19937 gen(time(0));
    for(unsigned int i = 0; i < p.skn_rows*32*32; i++){
       select_seed[i] = gen();
    }
    p.select_seed = (unsigned int*) cudaAllocAndCopy(select_seed, p.skn_rows*32*32*sizeof(unsigned int));

    double time = sketch_contruction(&p);
    std::cout << p.skn_rows << ";" << p.skn_cols << ";" << p.replicas << ";" << p.ts << ";" << p.ts*sizeof(unsigned int)*2*8 / (float) time << std::endl;

    return 0;
}
