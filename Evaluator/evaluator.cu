#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wignored-attributes"
#include <iostream>
#include <chrono>
#include <time.h>
#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_int_distribution.hpp>

#define CHUNK_SIZE (1024*1024*256)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ unsigned int parity(unsigned int x) {
    unsigned int y;
    y = x ^ (x >> 1);
    y = y ^ (y >> 2);
    y = y ^ (y >> 4);
    y = y ^ (y >> 8);
    y = y ^ (y >>16);
    return y & 1;
 }
 
 __device__ unsigned int nonlinear_h(unsigned int x) {
     return parity((x >> 0) | (x >> 1));
 }
 
 __device__ unsigned int is_set(unsigned int x, unsigned int pos) {
     return (x >> pos) & 1;
 }
 

 __device__ unsigned int h3(unsigned int x, unsigned int nsketches, unsigned int* seed) {
    unsigned int hash = 0;
    for(int i = 0; i < 32; i++){
        hash ^=  seed[i*nsketches]*is_set(x,i); 
    }
    return hash;
}


 __device__ int ech3(unsigned int v, unsigned int seed, unsigned int sbit){
     //First we compute the bitwise AND between the seed and the value
     int res = parity(v & seed) ^ nonlinear_h(v) ^ sbit ;
     //Aaaand here comes the parity
     return 2*res-1;
 }

 __global__ void construct_sketch(
    unsigned int skn_rows,
    unsigned int skn_cols,
    unsigned long n_values,
    unsigned int* __restrict__ key,
    unsigned int* __restrict__ value,
    unsigned int* __restrict__ select_seed,
    unsigned long long int* __restrict__ sketches_count,
    unsigned long long int* __restrict__ sketches_sum,
    unsigned int* __restrict__ sketches_min,
    unsigned int* __restrict__ sketches_max
) 
{
    unsigned int global_size = gridDim.x * blockDim.x;
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;


    for(unsigned long i = global_id; i < n_values; i += global_size){
        for(unsigned int r = 0; r < skn_rows; r++) {
            unsigned int select = 0;
            for(int k = 0; k < 32; k++)  if(is_set(key[i],k)) select ^= select_seed[r*32+k];
            select = select % skn_cols;

            //atomicAdd(&sketches_count[r*skn_cols+select], 1UL);
            //atomicAdd(&sketches_sum[r*skn_cols+select], (unsigned long) value[i]);
            //atomicMin(&sketches_min[r*skn_cols+select], 1);
            //atomicMax(&sketches_max[r*skn_cols+select], value[i]);
        }
    }
}

typedef struct{

    size_t skn_rows;
    size_t skn_cols;

    unsigned long long int* sk_cnt;
    unsigned long long int* sk_sum;
    unsigned int* sk_min;
    unsigned int* sk_max;

    unsigned long long int* gt_cnt;
    unsigned long long int* gt_sum;
    unsigned int* gt_min;
    unsigned int* gt_max;
    unsigned int* gt_keys;

    unsigned long ts;
    unsigned long nkeys;

    unsigned int* keys;
    unsigned int* values;
    unsigned int* select_seed;
    unsigned long long int* errors;

} parameters;

void* cudaAllocAndCopy(void* hst_ptr, size_t size){
    void* d_ptr;
    hipMalloc((void **) &d_ptr, size);
    hipMemcpy(d_ptr, hst_ptr, size, hipMemcpyHostToDevice);
    return d_ptr;
}

void writeSArrayToFile(const char* filename, int* elements, size_t size){
    FILE *f1 = fopen(filename, "w");
    assert(f1 != NULL);
    
    fwrite(elements, sizeof(int), size, f1);
    fclose(f1);
}

void* readMappedArrayFromFile(const char* filename, size_t * filesize = NULL){
    FILE *f1 = fopen(filename, "rb");
    assert(f1 != NULL);
    fseek(f1, 0, SEEK_END);
    size_t fsize1 = ftell(f1);
    if(filesize) *filesize=fsize1;
    fseek(f1, 0, SEEK_SET);
    void* tab1;
    hipHostAlloc(&tab1,fsize1, hipHostMallocMapped);
    size_t x = fread(tab1, fsize1, 1, f1);
    fclose(f1);

    return tab1;
}

void* readArrayFromFile(const char* filename, size_t * filesize = NULL){
    FILE *f1 = fopen(filename, "rb");
    assert(f1 != NULL);
    fseek(f1, 0, SEEK_END);
    size_t fsize1 = ftell(f1);
    if(filesize) *filesize=fsize1;
    fseek(f1, 0, SEEK_SET);
    void* tab1;
    hipHostMalloc(&tab1, fsize1);
    size_t x = fread(tab1, fsize1, 1, f1);
    fclose(f1);

    return tab1;
}

double sketch_contruction(parameters* p){
    size_t local = 64;
    int tot_SM = 0;
    hipDeviceGetAttribute(&tot_SM, hipDeviceAttributeMultiprocessorCount, 0);

    int occupancy = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&occupancy, construct_sketch, local, 0);

    size_t global = occupancy*tot_SM;

    auto begin = std::chrono::high_resolution_clock::now();
   
    construct_sketch<<<global/local, local>>>((unsigned int) p->skn_rows, (unsigned int) p->skn_cols, p->ts, p->keys, p->values, p->select_seed, p->sk_cnt, p->sk_sum, p->sk_min, p->sk_max);
    gpuErrchk(hipPeekAtLastError());

    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration_cast<std::chrono::milliseconds>(end-begin).count();
}


int main( int argc, const char* argv[] )
{
    parameters p;
    hipSetDevice(0);
    hipSetDeviceFlags(hipDeviceMapHost);

    p.skn_rows = (unsigned int) atoll(argv[1]);
    p.skn_cols = (unsigned int) atoll(argv[2]);

    
    //Initialize sketches
    hipMalloc((void **) &p.sk_cnt, p.skn_rows*p.skn_cols*sizeof(unsigned long long int));
    hipMemset(p.sk_cnt, 0, p.skn_rows*p.skn_cols*sizeof(unsigned long));

    hipMalloc((void **) &p.sk_sum, p.skn_rows*p.skn_cols*sizeof(unsigned long long int));
    hipMemset(p.sk_sum, 0, p.skn_rows*p.skn_cols*sizeof(unsigned long));

    hipMalloc((void **) &p.sk_min, p.skn_rows*p.skn_cols*sizeof(unsigned int));
    hipMemset(p.sk_min, 255, p.skn_rows*p.skn_cols*sizeof(unsigned int));

    hipMalloc((void **) &p.sk_max, p.skn_rows*p.skn_cols*sizeof(unsigned int));
    hipMemset(p.sk_max, 0, p.skn_rows*p.skn_cols*sizeof(unsigned int));

    hipMalloc((void **) &p.errors, 6*sizeof(unsigned long));
    hipMemset(p.errors, 0, 6*sizeof(unsigned long));

    size_t size = 0;
    p.keys = (unsigned int *) readMappedArrayFromFile(argv[3], &size);
    p.values = (unsigned int *)readMappedArrayFromFile(argv[4],&size);
    p.ts = size/sizeof(unsigned int);

    unsigned int* select_seed =  (unsigned int*) malloc(sizeof(unsigned int)*32*32*p.skn_rows);

    boost::random::mt19937 gen(time(0));
    for(unsigned int i = 0; i < p.skn_rows*32*32; i++){
       select_seed[i] = gen();
    }
    p.select_seed = (unsigned int*) cudaAllocAndCopy(select_seed, p.skn_rows*32*32*sizeof(unsigned int));

    double time = sketch_contruction(&p);
    std::cout << "construction_|;" << p.skn_rows << ";" << p.skn_cols << ";" << p.ts << ";" << p.ts*sizeof(unsigned int)*2*8 / (1000.0*1000.0*1000.0*time / 1000.0) << std::endl;

    hipHostFree(p.keys);
    hipHostFree(p.values);


    //If we made it until here, the sketch was created and we are ready for the accuracy experiments
    //p.gt_keys = (unsigned int*) readMappedArrayFromFile(argv[5], &size);
    //p.gt_cnt = (unsigned long long int*) readMappedArrayFromFile(argv[6]);
    //p.gt_sum = (unsigned long long int*) readMappedArrayFromFile(argv[7]);
    //p.gt_min = (unsigned int*) readMappedArrayFromFile(argv[8]);
    //p.gt_max = (unsigned int*) readMappedArrayFromFile(argv[9]);
    //p.nkeys = size/sizeof(unsigned int);

    //time = run_full_count_scan(&p);
    //std::cout << "Full count scan done" << std::endl;

    //time = run_groundtruth_scan(&p);
    //std::cout << "Groundtruh scan done" << std::endl;

    //Copy shit back
    //unsigned long long int* res = (unsigned long long int*) malloc(6*sizeof(unsigned long long int));
    //hipMemcpy(res, p.errors, 6*sizeof(long long int), hipMemcpyDeviceToHost);

    //std::cout << "Full reconstruction count error: " << (res[0]-res[1]) << std::endl; 
    //std::cout << "Correction " << (res[1]) << std::endl; 
    //std::cout << "Count error: " << (res[2]) << std::endl; 
    //std::cout << "Sum error: " << (res[3]) << std::endl; 
    //std::cout << "Min error: " << (res[4]) << std::endl; 
    //std::cout << "Max error: " << (res[5]) << std::endl; 
    //std::cout << p.skn_rows << ";" << p.skn_cols << ";" << (res[0]-res[1]) << ";" << res[2] << ";" << res[3] << ";" << res[4] << ";" << res[5] << std::endl;

    //hipHostFree(p.gt_keys);
    //hipHostFree(p.gt_cnt);
    //hipHostFree(p.gt_sum);
    //hipHostFree(p.gt_min);
    //hipHostFree(p.gt_max);
    
    return 0;
}
